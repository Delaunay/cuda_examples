
#include <hip/hip_runtime.h>
#include <cstdio>

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
    int width;
    int height;
    int stride;
    float* elements;
    bool cpu;
} Matrix;

Matrix make_cpu(int w, int h){
    Matrix m;
    m.width = w;
    m.height = h;
    m.elements = static_cast<float*>(malloc(w * h * sizeof(float)));
    m.cpu = true;
    return m;
}

Matrix make_gpu(int w, int h){
    Matrix m;
    m.width = w;
    m.height = h;
    auto size = w * h;
    hipMalloc(&m.elements, size);
    m.cpu = false;
    return m;
}

Matrix make_gpu_from(Matrix m){
    Matrix gpu = make_gpu(m.width, m.height);
    hipMemcpy(gpu.elements, m.elements, m.width * m.height, hipMemcpyHostToDevice);
    return gpu;
}

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col, float value)
{
    A.elements[row * A.stride + col] = value;
}

// Thread block size
#define BLOCK_SIZE 16

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
__device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements =
        &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];

    return Asub;
}

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

    Matrix d_B;
    d_B.width = d_B.stride = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = d_C.stride = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();

        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}


int main(){
    std::size_t size = 2048;

    Matrix A = make_cpu(size, size);
    Matrix B = make_cpu(size, size);
    Matrix C = make_cpu(size, size);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    float time = 0;

    for (int i = 0; i < 10; ++i){
        hipEventRecord(start, 0);
        for (int i = 0; i < 10; ++i){
            MatMul(A, B, C);
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

        float cuda_time = 0;
        hipEventElapsedTime(&cuda_time, start, stop);
        time += cuda_time;
    }

    printf("%f \n", time);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
